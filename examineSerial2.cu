//#include "examineSerial2.h"
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#define DOWNLIMIT 12
#define UPLIMIT 30
#define LAST_RANK 6
#define BITS_LINE 10
#define MALLOC_SIZE 240000 //80000 sugkrouseis
//#define KERNEL 512


void printTime(struct timespec,struct timespec);

void checker(char**);


__device__ float strToF(const char* s){
  float asd = 0, fact = 1;
  if (*s == '-'){
    s++;
    fact = -1;
  };
  int ch=0;
  for ( ch = 0; *s; s++){
    if (*s == '.'){
      ch = 1; 
      continue;
    };
    int d = *s - '0';
    if (d >= 0 && d <= 9){
      if (ch) fact /= 10.0F;
      asd = asd * 10.0F + (float)d;
    };
  };
  return asd * fact;
}

/**
 *checkerSer
 *checks the limit of the cordinates
 *checks the limit of the time
 *starts the clock and then goes to the for
 *it takes 3 lines each time in order to check the coordinates
 *it check if there are in the space we need
 *then it stops the clock
 *and checks if the time limit is exided
 *if yes it just itterate to the end
 *if not it continues till the end
 *then it returns the usable coordinates
 */
__global__ void checkerSer(char* buffer, long * bufferSize, int * usableCoordinates)	{

	//printf("%d\n",coordinateNumberToExamine);
	//char line[30]; /* or other suitable maximum line size */
	float distance=0;
	//int a;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	bufferSize[0] = bufferSize[0]/10;
	if(idx<bufferSize[0]/3) {
//		for(a = 0; a < bufferSize/10; a=a+3 ) {		
		char nLine1[BITS_LINE];
		char nLine2[BITS_LINE];
		char nLine3[BITS_LINE];
		float value[3];
		int b;
		for(b=0 ; b < (10); b++) {
			nLine1[b] = buffer[idx*3*10+(10*0)+b];
			nLine2[b] = buffer[idx*3*10+(10*1)+b];
			nLine3[b] = buffer[idx*3*10+(10*2)+b];			
		}
		nLine1[9] = '\0';
		nLine2[9] = '\0';
		nLine3[9] = '\0';
		value[0] = strToF(nLine1);
		value[1] = strToF(nLine2);
		value[2] = strToF(nLine3);
		distance= sqrtf(value[0]*value[0] + value[1]*value[1] + value[2]*value[2]);
		if((distance>=DOWNLIMIT) && (distance<=UPLIMIT)) {
//			__syncthreads();
			//printf("%f %f %f",value[0],value[1],value[2]);
			atomicAdd(usableCoordinates,1);
			//(*usableCoordinates)++;
			//(*uc)++;
		}
//__syncthreads();
		//distance=0;
	}
}


/*
* char *argv[] parameters:
*  argv[1]: max limit to examine or -1 to mark no limit
*  argv[2]: max limit time to examine coordinates or -1 to mark no limit
*  argv[3]: File's name
*  argv[4]: Thread that are being used or -1 to mark no limit
*  argv[5]: Proccesses to use MPI or -1 to mark no limit
*/

/**main
 *
 *This function is the main which calls the other to work
 *has 2 variables for input argc and argv[]
 *argc is the number of the argv cells
 *argv values are explained above
 *So main checks the number of argv values and if its ok it
 *start working.Specificaly it start's the clock
 *it calls checker function using the input 
 *file (created  with the generator)
 *and it end's the time after that(calls printTime function).
 */
int main (int argc, char *argv[]) {
	if(argc!=6) {
		printf("Wrong number of arguents");
		exit(0);
	}
	if(atof(argv[1])==0||atof(argv[2])==0||atof(argv[4])==0||atof(argv[5])==0){
		printf("Arguments cant be 0\n");
		exit(0);
	}
	// getting the start time of program
	struct timespec startTime, endTime;
	clock_gettime(CLOCK_MONOTONIC, &startTime);

	checker(argv);

	// getting the end time of program
	clock_gettime(CLOCK_MONOTONIC, &endTime);
	printTime(startTime,endTime);

	return 0;
}

/**printTime
 *
 *print time use's two timespec struct variables
 *whitch contains a value that express the time of
 *the system and by doing the right maths
 *it calculates the time between the two timespecs in seconds
 *It is used by main in order to print the time the program was running
 */
void printTime(struct timespec a,struct timespec b) {
	const int DAS_NANO_SECONDS_IN_SEC = 1000000000;
	long timeElapsed_s = b.tv_sec - a.tv_sec;
	long timeElapsed_n = b.tv_nsec - a.tv_nsec;
	if(timeElapsed_n < 0) {
		timeElapsed_n = DAS_NANO_SECONDS_IN_SEC + timeElapsed_n;
		timeElapsed_s--;
	}
	printf("Time: %ld.%09ld secs \n", timeElapsed_s, timeElapsed_n);
}

/**
 *checker
 * opens the data file
 * find it's size and then goes back to start
 * it reads it using the buffer and prints the errors if there are any
 * then it close the file
 * and then calls the checkerSer in order to find the usable coordinates
 * when checkerSer ends it prints the number of the usable coordinates
 */
void checker(char *argv[]) {

	/*
	int coordinatesToExamine = atoi(argv[1]);
	if(coordinatesToExamine > 0) {
		if(coordinatesToExamine < size) {
			size = coordinatesToExamine;
		}
	}*/

	// reading the values from the file
	FILE * pFile;
	long lSize, lSizeF;
	long lSizeL = 0;
	char * buffer;
	long bufferSize;
	int usableCoordinates = 0;	
	int breaker = 2;
        //int  h_usableCoordinates = 0;  

	pFile = fopen ( argv[3] , "r" );
	if (pFile==NULL) {
		printf("File error");
		exit (1);
	}

	// obtain file size:
	fseek (pFile , 0 , SEEK_END);
	lSizeF = ftell (pFile);
	int coordinatesToExamine = atoi(argv[1]);
	if(coordinatesToExamine > 0) {
		if(coordinatesToExamine*30 < lSizeF) {
			lSizeF = coordinatesToExamine*30;
		}
	}
	//printf("%ld\n", lSizeF);
	rewind (pFile);
	if(lSizeF > MALLOC_SIZE) {
		lSize = MALLOC_SIZE;
	} else {
		lSize = lSizeF;
		breaker--;
	}

	do {
		bufferSize = lSize - lSizeL;
		// allocate memory to contain the whole file:

		buffer = (char*) malloc (sizeof(char)*bufferSize+1);
		if (buffer == NULL) {
			printf("Memory error");
			exit (2);
		}
		// copy the file into the buffer:
		bufferSize = fread (buffer, 1, bufferSize, pFile);
		buffer[bufferSize] = '\0';
		/* the whole file is now loaded in the memory buffer. */

//######################
		char * d_buffer;
		long * d_bufferSize;
		long * h_bufferSize;
		int * d_usableCoordinates;
		h_bufferSize = &bufferSize;
		//h_usableCoordinates = (int *)malloc(sizeof(int));        // Allocate array on host
		if(hipMalloc((void**)&d_buffer, bufferSize) != hipSuccess) {
			printf("hipMalloc error 1\n");
			exit(4);
		}
		if(hipMalloc((void**)&d_bufferSize, sizeof(long)) != hipSuccess) {
			printf("hipMalloc error 2\n");
			free(d_buffer);
			exit(4);
		}
		if(hipMalloc((void**)&d_usableCoordinates, sizeof(int)) != hipSuccess) {
			printf("hipMalloc error 3\n");
			free(d_buffer);
			free(d_bufferSize);
			exit(4);
		}
		if(hipMemcpy(d_buffer,buffer, bufferSize, hipMemcpyHostToDevice) != hipSuccess) {
			printf("hipMemcpy error 4\n");
			hipFree(d_buffer);
			hipFree(d_bufferSize);
			hipFree(d_usableCoordinates);
			exit(5);
		}
		if(hipMemcpy(d_bufferSize,h_bufferSize, sizeof(long), hipMemcpyHostToDevice) != hipSuccess) {
			printf("hipMemcpy error 5\n");
			hipFree(d_buffer);
			hipFree(d_bufferSize);
			hipFree(d_usableCoordinates);
			exit(5);
		}
		if(hipMemcpy(d_usableCoordinates,&usableCoordinates,sizeof(int),hipMemcpyHostToDevice)!=hipSuccess){
			printf("hipMemcpy error 7\n");
			hipFree(d_buffer);
			hipFree(d_bufferSize); 
                        hipFree(d_usableCoordinates);
			exit(5);
		}

//*****************Kernel Run*******************************//
		//checkerSer<<<bufferSize/(30*KERNEL) + 1, KERNEL>>>(d_buffer, d_bufferSize, d_usableCoordinates);
		long loula = bufferSize/30;
		int KERNEL = 512;
		if (loula < KERNEL) {
			KERNEL = loula;
		}		
		dim3 grid(loula/KERNEL + (loula%KERNEL == 0 ? 0:1));            // defines a grid of 256 x 1 x 1 blocks
		dim3 block(KERNEL);       // defines a block of 512 x 512 x 1 threads
		checkerSer<<<grid, block>>>(d_buffer, d_bufferSize, d_usableCoordinates);
	//	checkerSer<<<1,512>>>(d_buffer, d_bufferSize, d_usableCoordinates);
		if(hipMemcpy(&usableCoordinates,d_usableCoordinates, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
			printf("hipMemcpy error 6\n");
			hipFree(d_buffer);
			hipFree(d_bufferSize);
			hipFree(d_usableCoordinates);
			exit(5);
		}
//######################
//printf("%d \n", usableCoordinates);
//		h_usableCoordinates = usableCoordinates + h_usableCoordinates;
//printf("Line 290\n");		

		hipFree(d_buffer);
		hipFree(d_bufferSize);
		hipFree(d_usableCoordinates);
//usableCoordinates = *d_usableCoordinates;
		free (buffer);
		lSizeL = lSize;
		lSize = lSize + MALLOC_SIZE;
		//printf("Loula: %d, %ld %ld\n", breaker, lSize, lSizeL);
		if( lSize >= lSizeF ) {
			lSize = lSizeF;
			breaker--;
		}
	} while(breaker > 0);
	fclose (pFile);
//	printf("add the numbers number %d \n",h_usableCoordinates);
	printf("Number of usable cordinates = %d\n", usableCoordinates);
}
